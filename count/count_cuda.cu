
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <math.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__global__
void countElem(int n, int find, int *data, int *d_result)
{
  int batch_size = n/gridDim.x;
  int idx = threadIdx.x + batch_size*blockIdx.x;

  int batch_count = 0;

  for(int i = idx; i<batch_size*(blockIdx.x+1); i+=blockDim.x){
    if(data[i] == find){
      batch_count++;
    }
  }
 
  atomicAdd(&d_result[blockIdx.x], batch_count);
}

int countElemCPU(int n, int find, int *data){
  unsigned int total_count = 0;
  for(int i = 0; i<n; i++){
    if(data[i] == find){
      total_count++;
    }
  }
  return total_count;
}

__global__
void initialize(int* data, int N){

  int idx = threadIdx.x+blockDim.x*blockIdx.x;

  for(int i = idx; i <  N; i += blockDim.x){
    data[i] = 50;
  }
}

int main(void)
{

  auto clock = std::chrono::high_resolution_clock();
  auto start = clock.now();

  int deviceCount;
  HANDLE_ERROR(hipGetDeviceCount(&deviceCount));

  if(deviceCount != 4 && deviceCount != 2){
    std::cerr << "GPUs available are: " << deviceCount << std::endl;
    if(deviceCount != 1){
      std::cerr << "Device count not suitable." << std::endl;
      return 1;
    }
  }

  int N = 1<<28;
  int blockSize = 256;
  int numBlocks = 8;

  int batchN = N/deviceCount;

  for(int gpuID = 0; gpuID<deviceCount; gpuID++){
    int *d_result = new int[numBlocks];
    int *result = new int[numBlocks];
    int *data = new int[batchN];
    int *d_data = new int[batchN];

    //In this case we initialize the data so it's not necessary to divide the data

    hipStream_t stream;
    hipStreamCreate(&stream);

    HANDLE_ERROR(hipSetDevice(gpuID));
    HANDLE_ERROR(hipMallocAsync(&d_data, batchN*sizeof(int), stream));
    HANDLE_ERROR(hipMallocAsync(&d_result, numBlocks*sizeof(int), stream));

    initialize<<<numBlocks, blockSize>>>(d_data, batchN);
  
    countElem<<<numBlocks, blockSize>>>(batchN, 50,d_data, d_result);

    HANDLE_ERROR(hipMemcpy(result, d_result, numBlocks*sizeof(int), hipMemcpyDeviceToHost));

    int final_count = 0;
    for(int i = 0; i<numBlocks; i++){
      final_count += result[i];
    }

    std::cout << "Element count: " << batchN << std::endl;  
    std::cout << "Device variable value: " << final_count <<std::endl;

    // Free memory
    HANDLE_ERROR(hipFreeAsync(d_result, stream));
    HANDLE_ERROR(hipFreeAsync(d_data, stream));

    auto end = clock.now(); 
    long long microseconds = std::chrono::duration_cast<std::chrono::microseconds>(end-start).count();
    std::cout << "Time: "<<  (float) microseconds/1000 << "ms" << std::endl;
  }

  HANDLE_ERROR(hipDeviceSynchronize());

  //Introduce MPI communication to send the result and merge it

  return 0;
}